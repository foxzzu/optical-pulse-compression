#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_complex.h"
#include "fft_naive.h"

#define SIZE	(int)pow(2,10) //I can do 32 threads per block BUT I am not sure how exactly this number works

#ifndef M_PI
#define M_PI acos(-1.0)
#endif

int main()
{
	hipDoubleComplex * original_vector = (hipDoubleComplex *)calloc(SIZE, sizeof(hipDoubleComplex));
	for (int j = 0; j < SIZE; j++)
		original_vector[j] = make_hipDoubleComplex(j + 1, 0);

	hipDoubleComplex * d_init_vec;
	hipMalloc(&d_init_vec, sizeof(hipDoubleComplex)); //initialise the CUDA subsystem

	hipFree(d_init_vec);

	original_vector = fft_wrapper(original_vector, SIZE);

	printf("done");
	hipFree(original_vector);
	return 0;
}
